#include "hip/hip_runtime.h"
/*
	File: gpu.cu
    Author(s): 
		Austin Erck - University of the Pacific, ECPE 251, Spring 2021
	Description:
    	This program implements Shi Tomasi Feature Detection using NVIDIA's CUDA framework. 
*/

#include <argp.h>
#include "image_template.h"

#define RADIUS_OF_FEATURE_MARKER 8

int main(int argc, char **argv){

	// Handle arguments
	// TODO: Read from argv
	char *filepath = NULL;
	uint8_t verbosity = 0; // Determines how much information should be shown
	float sigma = 1.1; // Sigma of the gaussian distribution
	uint64_t blockSize = 16; // CUDA block size
	uint64_t windowSize = 4; // Size of a pixel 'neighborhood'
	float sensitivity = 0.1; // Number of features = sensitivity*image_width

	// Setup timers
    struct timeval computationStart, computationEnd;

	// Setup CUDA pointers
	float *h_data1; //host pointers
	float *d_data1; //device pointers

	// Read image into first data array
	int width = 0, height = 0;
	//const float &initialImage = h1_data;
	read_image_template(filepath, &h_data1, &width, &height); // h_data1 = initialImage

	// Generate kernels
	int GWidth, dGWidth;
	//gaussian(sigma, &h_G, &GWidth);
	//gaussianDerivative(sigma, &h_dG, &dGWidth);

	// Malloc data on devices
	hipMalloc((void **)&d_data1, sizeof(float) * width * height);

	// Begin computation timer
    gettimeofday(&compStart, NULL);

	// Populate data on devices
	hipMemcpy(d_data1, h_data1, sizeof(float) * width * height, hipMemcpyHostToDevice);

	// Setup CUDA grid and blocks
    dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(width/blockSize, height/blockSize); // ASSUMPTION: Image is divisible by 16

	// TODO: Image processing

	// Copy data from device to host
	hipMemcpy(h_data1, d_data1, sizeof(float) * width * height, hipMemcpyDeviceToHost);

	// Sync CUDA threads and measure computation time
    hipDeviceSynchronize();
    gettimeofday(&compEnd, NULL);

    // Save output image to disk
    const char outputFilename[] = "corners.pgm";
    write_image_template(outputFilename, h_data1, width, height);

    // Free data from host and devices
    free(h_data1);
    hipFree(d_data1);

    // Print benchmarching information
	printf("%d, %f, %x, %x, %f, %Lf\n", width, sigma, blockSize, windowSize, sensitivity, calculateTime(computationStart, computationEnd));

	return 0;
}

long double calculateTime(struct timeval start, struct timeval end) {
	return (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);	
}

void gaussian(const float sigma, float** kernel, int* width) {
    
    float a = roundf(2.5 * sigma - 0.5);
    float w = 2 * a + 1;
    float* G = (float*)malloc(w * sizeof(float));
    float sum = 0;

    //printf("G_w: %f\n", w);
    int i;
    for(i = 0; i < w; i++) {
        G[i] = expf(-1.0 * powf((float)(i) - a, 2.0) / (2.0 * powf(sigma, 2.0)));
        sum += G[i];
    }

    // Normalize gaussian kernel peaks
    for(i = 0; i < w; i++) {
        G[i] = G[i] / sum;
        //printf("G[%d]: %f\n", i, G[i]);
    }

    // Set results to pointers
    *kernel = G;
    *width = (int)w;
}

void gen_kernel(float *gkernel, float *dkernel, float sigma, int a, int w){
	a = (int)round(2.5 * sigma -.5);
	kernel_width = 2 * a + 1;
	
	float sum_gkern = 0;
	float sum_dkern = 0;

	int i;
	for(i = 0; i < w; i++){
		gkernel[i] = (float)exp( (float)(-1.0 * (i-a) * (i-a)) / (2 * sigma * sigma));

		dkernel[i] = (-1 * (i - a)) * );
		 G[i] = expf(-1.0 * );
		sum_gkern = sum_gkern + gkernel[i];
		sum_dkern = sum_dkern - (float)i * dkernel[i];
	}

	//reverse the kernel by creating a new kernel, yes not ideal
	float *newkernel = (float *)malloc(sizeof(float) * w);
	for (i = 0; i < w; i++){
		dkernel[i] = dkernel[i] / sum_dkern;
		gkernel[i] = gkernel[i] / sum_gkern;
		newkernel[w-i] = dkernel[i];
	}

	//copy new kernel back in
	for (i = 0; i < w; i++){
		dkernel[i] = newkernel[i+1];
	}
	free(newkernel);
}