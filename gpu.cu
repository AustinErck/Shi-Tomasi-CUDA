#include "hip/hip_runtime.h"
/*
	File: gpu.cu
    Author(s): 
		Austin Erck - University of the Pacific, ECPE 251, Spring 2021
	Description:
    	This program implements Shi Tomasi Feature Detection using NVIDIA's CUDA framework. 
*/

#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <argp.h>
#include <hip/hip_runtime.h>
#include "image_template.h"
#include "gpu.h"

#define RADIUS_OF_FEATURE_MARKER 8

int main(int argc, char **argv){

	// Handle arguments
	// TODO: Read from argv
	char *filepath = NULL;
	int verbosity = 0; // Determines how much information should be shown
	float sigma = 1.1; // Sigma of the gaussian distribution
	int blockSize = 16; // CUDA block size
	int windowSize = 4; // Size of a pixel 'neighborhood'
	float sensitivity = 0.1; // Number of features = sensitivity*image_width

	// Setup timers
    struct timeval computationStart, computationEnd;

	// Setup CUDA pointers
	float *h_data1, *h_G, *h_DG; //host pointers
	float *d_data1, *d_data2, *d_data3, *d_G, *d_DG; //device pointers

	// Read image into first data array
	int width = 0, height = 0;
	//const float &initialImage = h1_data;
	read_image_template(filepath, &h_data1, &width, &height); // h_data1 = initialImage

	// Calculate constants
	const int bytesPerImage = sizeof(float) * width * height;
	const int bytesPerBlock = sizeof(float) * blockSize * blockSize;

	// Setup CUDA grid and blocks based on image size
	dim3 dimBlock(blockSize, blockSize);
	dim3 dimGrid(width/blockSize, height/blockSize); // ASSUMPTION: Image is divisible by 16

	// Generate kernels
	int kernelWidth;
	generateKernels(h_G, h_DG, &kernelWidth, sigma);

	// Malloc data on devices
	hipMalloc((void **)&d_data1, bytesPerImage);
	hipMalloc((void **)&d_data2, bytesPerImage);
	hipMalloc((void **)&d_data3, bytesPerImage);
	hipMalloc((void **)&d_G, sizeof(float) * kernelWidth);
	hipMalloc((void **)&d_DG, sizeof(float) * kernelWidth);

	// Begin computation timer
    gettimeofday(&computationStart, NULL);

	// Populate data on devices from host
	hipMemcpy(d_data1, h_data1, bytesPerImage, hipMemcpyHostToDevice);
	hipMemcpy(d_G, h_G, sizeof(float) * kernelWidth, hipMemcpyHostToDevice);
	hipMemcpy(d_DG, h_DG, sizeof(float) * kernelWidth, hipMemcpyHostToDevice);

	// Temp Horizontal/Vertical convolutions
	convolve<<<dimGrid,dimBlock, bytesPerBlock>>>(d_data1, d_data2, width, height, d_G, 1, kernelWidth); // data1 = temp_horizontal
    convolve<<<dimGrid,dimBlock, bytesPerBlock>>>(d_data1, d_data3, width, height, d_G, kernelWidth, 1); // data2 = temp_vertical
   
    // Horizontal/Vertical convolutions
    convolve<<<dimGrid,dimBlock, bytesPerBlock>>>(d_data2, d_data1, width, height, d_DG, kernelWidth, 1); // data1 = horizontal
    convolve<<<dimGrid,dimBlock, bytesPerBlock>>>(d_data3, d_data2, width, height, d_DG, 1, kernelWidth); // data2 = vertical

	// Compute eigen values
	computeEigenValues<<<dimGrid,dimBlock, bytesPerBlock * 2>>>(d_data1, d_data2, d_data3, width, height, windowSize); // d_data3 = eigenValues

	// TODO: Find features

	// Copy data from device to host
	hipMemcpy(h_data1, d_data1, bytesPerImage, hipMemcpyDeviceToHost);

	// Sync CUDA threads and measure computation time
    hipDeviceSynchronize();
    gettimeofday(&computationEnd, NULL);

    // Save output image to disk
    char outputFilename[] = "corners.pgm";
    write_image_template(outputFilename, h_data1, width, height);

    // Free data from host and devices
    free(h_data1);
    hipFree(d_data1);

    // Print benchmarching information
	printf("%d, %f, %x, %x, %f, %Lf\n", width, sigma, blockSize, windowSize, sensitivity, calculateTime(computationStart, computationEnd));

	return 0;
}

long double calculateTime(struct timeval start, struct timeval end) {
	return (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);	
}

void generateKernels(float* G, float* DG, int* width, const float sigma){

	// Calculate a and w(idth) variables used in guassian and derivative guassian kernel calculations
	float a = roundf(2.5 * sigma - 0.5);
    int w = 2 * a + 1;

	// Track total of all values used in each kernel
    float sumG = 0, sumDG = 0;

	// Loop through the width of the kernel and populate kernels while calculating the sum of each
	int i;
	for(i = 0; i < w; i++) {
        G[i] = expf(-1.0 * powf((float)(i) - a, 2.0) / (2.0 * powf(sigma, 2.0)));
		DG[i] = -1.0 * ( (float)(i + 1) - 1.0 - a) * expf(-1.0 * powf((float)(i + 1) - 1.0 - a, 2.0) / (2.0 * powf(sigma, 2.0)));
        sumG += G[i];
		sumDG -= i * DG[i];
    }

	// Divide each value in the kernel by the total sum of the kernel
	for (i = 0; i < w; i++){
		G[i] = G[i] / sumG;
		DG[i] = DG[i] / sumDG;
	}

	// Flip derivative kernel
	for (i = 0; i < w/2; i++) {
        const float temp = DG[w - i - 1];
        DG[w - i - 1] = DG[i];
        DG[i] = temp;
    }
}

__global__
void convolve(const float* image, float* outputImage, const int imageWidth, const int imageHeight, const float* kernel, const int kernelWidth, const int kernelHeight) {

    // Calculate kernel center constants
    const int kernelCenterX = kernelWidth / 2;
    const int kernelCenterY = kernelHeight / 2;
    
    // Set initial pixel value to zero
    float sum = 0;

    // Get x and y based on thread and block index
    const int xBlockOffset = blockIdx.x * blockDim.x;
    const int yBlockOffset = blockIdx.y * blockDim.y;
    const int xLocal = threadIdx.x;
    const int yLocal = threadIdx.y;
    const int xGlobal = xLocal + xBlockOffset;
	const int yGlobal = yLocal + yBlockOffset;

    // Setup shared data array
    extern __shared__ float blockData[];
    blockData[yLocal * blockDim.x + xLocal] = image[yGlobal * imageWidth + xGlobal];
    __syncthreads();

    // Loop through each pixel of the   kernel
    int i, j;
    for(j = 0; j < kernelHeight; j++) {
        for(i = 0; i < kernelWidth; i++) {
        
            // Calculate offset based on current pixel in kernel
            int xCalculated = xGlobal + (i - kernelCenterX);
            int yCalculated = yGlobal + (j - kernelCenterY);
            
            // Check that pixel is not out of bounds
            if(xCalculated < 0 || xCalculated >= imageWidth || yCalculated < 0 || yCalculated >= imageHeight) {
                continue;
            }

            // Add image value multipled by kernel value to sum
			// Both cases perform the same action, however if possible local data(blockData) is used instead of global data(image)
            if(xCalculated >= xBlockOffset && xCalculated < xBlockOffset + blockDim.x && yCalculated >= yBlockOffset && yCalculated < yBlockOffset + blockDim.y) {
                // Go to block data

                // Calculate part of the convolve value based on image and kernel pixel.
                sum += kernel[j * kernelWidth + i] * blockData[(yCalculated - yBlockOffset) * blockDim.x + (xCalculated - xBlockOffset)];
            } else {
                // Go to global data

                // Calculate part of the convolve value based on image and kernel pixel.
                sum += kernel[j * kernelWidth + i] * image[yCalculated * imageWidth + xCalculated];
            }
        }
    }

    // Write sum to memory
    outputImage[yGlobal * imageWidth + xGlobal] = sum;
}

__global__
void computeEigenValues(const float* horizontalImage, const float* verticalImage, float* eigenValues, const int imageWidth, const int imageHeight, const int windowSize) {

	// Calculate window center constant
    const int windowCenter = windowSize / 2;
    
    // Set initial pixel value to zero
    float sumIXX = 0, sumIYY = 0, sumIXIY = 0;

    // Get x and y based on thread and block index
    const int xBlockOffset = blockIdx.x * blockDim.x;
    const int yBlockOffset = blockIdx.y * blockDim.y;
    const int xLocal = threadIdx.x;
    const int yLocal = threadIdx.y;
    const int xGlobal = xLocal + xBlockOffset;
	const int yGlobal = yLocal + yBlockOffset;

    // Setup shared data array
    extern __shared__ float sharedData[];
	float *horizontalImageLocal = (float*)&sharedData; // Use first half of shared memory for horizontal image
	float *verticalImageLocal = (float*)&sharedData + (sizeof(float) * imageWidth * imageHeight); // Use second half of shared memory for vertical image
    horizontalImageLocal[yLocal * blockDim.x + xLocal] = horizontalImage[yGlobal * imageWidth + xGlobal];
	verticalImageLocal[yLocal * blockDim.x + xLocal] = verticalImage[yGlobal * imageWidth + xGlobal];
    __syncthreads();

    // Loop through each pixel of the   kernel
    int i, j;
    for(j = 0; j < windowSize; j++) {
        for(i = 0; i < windowSize; i++) {
        
            // Calculate offset based on current pixel in kernel
            const int xCalculated = xGlobal + (i - windowCenter);
            const int yCalculated = yGlobal + (j - windowCenter);
            
            // Check that pixel is not out of bounds. Skip if it is
            if(xCalculated < 0 || xCalculated >= imageWidth || yCalculated < 0 || yCalculated >= imageHeight) {
                continue;
            }

            // Determine sum values for ixx, iyy, and ixiy
			// Both cases perform the same action, however if possible local data(horizontalImageLocal & verticalImageLocal) is used instead of global data(horizontalImage & verticalImage)
            if(xCalculated >= xBlockOffset && xCalculated < xBlockOffset + blockDim.x && yCalculated >= yBlockOffset && yCalculated < yBlockOffset + blockDim.y) {
                // Go to local data

				// Calculate array offset
				const int arrayOffset = (yCalculated - yBlockOffset) * blockDim.x + (xCalculated - xBlockOffset);

                // Calculate part of the convolve value based on image and kernel pixel.
                sumIXX += powf(horizontalImageLocal[arrayOffset], 2.0); // horizontalImage^2
				sumIYY += powf(verticalImageLocal[arrayOffset], 2.0); // verticalImage^2
				sumIXIY += horizontalImageLocal[arrayOffset] * verticalImageLocal[arrayOffset]; // horizontalImage * verticalImage
            } else {
                // Go to global data

				// Calculate part of the convolve value based on image and kernel pixel.
                sumIXX += powf(horizontalImage[yCalculated * imageWidth + xCalculated], 2.0); // horizontalImage^2
				sumIYY += powf(verticalImage[yCalculated * imageWidth + xCalculated], 2.0); // verticalImage^2
				sumIXIY += horizontalImage[yCalculated * imageWidth + xCalculated] * verticalImageLocal[yCalculated * imageWidth + xCalculated]; // horizontalImage * verticalImage
            }
        }
    }

	// Calculate eigen values
	const float temp1 = (sumIXX + sumIYY)/2;
	const float temp2 = powf( powf(sumIXX + sumIYY, 2.0)/4.0 - (sumIXX * sumIYY - powf(sumIXIY, 2.0)), 0.5);
	float eigenValue1 = temp1 + temp2;
	float eigenValue2 = temp1 - temp2;

	// Save smaller of the two eigen values
	eigenValues[j * imageWidth + i] = (eigenValue1 >= eigenValue2) ? eigenValue2 : eigenValue1;
}
